#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath> // For fabs and cos
#include "hip/hip_runtime_api.h"

// The kernel remains the same, it just processes the data it's given.
__global__ void do_work(double *data, int N, int offset) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + offset;
    if (i < N) {
        for (int j = 0; j < 20; j++) {
            data[i] = cos(data[i]);
            data[i] = sqrt(fabs(data[i]));
        }
    }
}

int main()
{
    // Total data size (approx 1GB)
    int total_data = 1 << 27;
    double *d_data;
    double *h_data;

    // STEP 1: Use pinned (page-locked) host memory for faster, asynchronous transfers.
    // This is a prerequisite for overlapping memory copies with kernel execution.
    checkCudaErrors(hipHostMalloc((void**)&h_data, total_data * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&d_data, total_data * sizeof(double)));

    // Initialise host data with random values
    srand(0);
    for (int i = 0; i < total_data; i++) {
        h_data[i] = (double)rand() / (double)RAND_MAX;
    }

    // STEP 2: Define the number of streams (chunks) to break the work into.
    int num_streams = 8;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamCreate(&streams[i]));
    }

    // Set up CUDA events for accurate timing
    float time;
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));

    int blocksize = 256;
    // Calculate the size of each data chunk
    int chunk_size = total_data / num_streams;

    // STEP 3: Loop through the streams, issuing asynchronous operations.
    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        
        // Calculate the number of blocks needed for this specific chunk
        int nblocks = (chunk_size - 1) / blocksize + 1;

        // Asynchronously copy a chunk of data from Host to Device in its specific stream.
        // The CPU does not wait for this to complete.
        checkCudaErrors(hipMemcpyAsync(&d_data[offset], &h_data[offset],
                                       chunk_size * sizeof(double),
                                       hipMemcpyHostToDevice, streams[i]));

        // Launch the kernel to process the data chunk in the same stream.
        // This kernel launch is queued after the copy in the same stream.
        // It will only execute after the HtoD copy for this stream is complete.
        do_work<<<nblocks, blocksize, 0, streams[i]>>>(d_data, total_data, offset);

        // Asynchronously copy the processed chunk from Device to Host in the same stream.
        // This is queued after the kernel. It will execute after the kernel for this stream is done.
        checkCudaErrors(hipMemcpyAsync(&h_data[offset], &d_data[offset],
                                       chunk_size * sizeof(double),
                                       hipMemcpyDeviceToHost, streams[i]));
    }

    // Wait for all operations in all streams to complete before stopping the timer.
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&time, start, stop));
    printf("Total processing time with streaming: %g ms\n", time);

    // Clean up streams and memory
    for (int i = 0; i < num_streams; i++) {
        checkCudaErrors(hipStreamDestroy(streams[i]));
    }
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipHostFree(h_data)); // Use hipHostFree for pinned memory

    return EXIT_SUCCESS;
}
